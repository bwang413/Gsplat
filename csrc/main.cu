#include <iostream>
#include <cstring>

#include "forward.h"


int main() {
    int num_points = 24;

    int num_mean = num_points * 3;
    int num_scale = num_points * 3;
    int num_quat = num_points * 4;
    int num_cov3d = num_points * 6;
    int num_cov2d = num_points * 3;
    int num_view = 16;

    float *means = new float[num_mean];
    float *scales = new float[num_scale];
    float *quats = new float[num_quat];
    float *covs3d = new float[num_cov3d];
    float *covs2d = new float[num_cov2d];
    float viewmat [] = {
        1.f, 0.f, 0.f, 0.f,
        0.f, 1.f, 0.f, 0.f,
        0.f, 0.f, 1.f, -1.f,
        0.f, 0.f, 0.f, 1.f
    };

    for (int i = 0; i < num_points; ++i) {
        // float x [] = {(float) i * 0.2, (float) i * 0.4, (float) i * 0.6};
        float x [] = {0.f, 0.f, 0.f};
        float s [] = {(float) i + 0.1f, (float) i + 0.1f, (float) i + 0.1f};
        float q [] = {1.f, 0.f, 0.f, 0.f};
        std::memcpy(&means[3 * i], &x, sizeof(float) * 3);
        std::memcpy(&scales[3 * i], &s, sizeof(float) * 3);
        // printf("scales %d, %.2f, %.2f, %.2f\n", i, scales[3*i], scales[3*i+1], scales[3*i+2]);
        std::memcpy(&quats[4 * i], &q, sizeof(float) * 4);
        // printf("quats %d, %.2f, %.2f, %.2f, %.2f\n", i, quats[4*i], quats[4*i+1], quats[4*i+2], quats[4*i+3]);
    }

    float *scales_d, *means_d, *quats_d, *viewmat_d, *covs3d_d, *covs2d_d;

    hipMalloc((void**) &scales_d, num_scale * sizeof(float));
    hipMalloc((void**) &means_d, num_mean * sizeof(float));
    hipMalloc((void**) &quats_d, num_quat * sizeof(float));
    hipMalloc((void**)&viewmat_d, num_view * sizeof(float));

    hipMemcpy(scales_d, scales, num_scale * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(means_d, means, num_mean * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(quats_d, quats, num_quat * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(viewmat_d, viewmat, num_view * sizeof(float), hipMemcpyHostToDevice);

    // allocate memory for outputs
    hipMalloc((void**)&covs3d_d, num_cov3d * sizeof(float));
    hipMalloc((void**)&covs2d_d, num_cov2d * sizeof(float));

    project_gaussians_forward_impl(num_points, means_d, scales_d, 1.f, quats_d, viewmat_d, covs3d_d, covs2d_d);
    hipMemcpy(covs3d, covs3d_d, num_cov3d * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(covs2d, covs2d_d, num_cov2d * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < num_points; ++i) {
        printf("covs3d %d ", i);
        for (int j=0; j < 6; ++j) {
            printf("%.2f,", covs3d[6*i+j]);
        }
        printf("\n");

        printf("covs2d %d ", i);
        for (int j=0; j < 3; ++j) {
            printf("%.2f,", covs2d[3*i+j]);
        }
        printf("\n");
    }

    hipFree(scales_d);
    hipFree(quats_d);
    hipFree(covs3d_d);

    delete[] scales;
    delete[] means;
    delete[] quats;
    delete[] covs3d;
    delete[] covs2d;
    return 0;
}
