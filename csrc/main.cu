#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <math.h>

#include "forward.cuh"
#include "tgaimage.h"


int main() {
    int num_points = 24;
    const float fov_x = M_PI / 2.f;
    const int W = 128;
    const int H = 128;
    const float focal = 0.5 * (float) W / tan(0.5 * fov_x);
    const dim3 tile_bounds = {(W + BLOCK_X - 1) / BLOCK_X, (H + BLOCK_Y - 1) / BLOCK_Y, 1};
    const dim3 img_size = {W, H, 1};
    const dim3 block = {BLOCK_X, BLOCK_Y, 1};

    int num_cov3d = num_points * 6;
    int num_view = 16;

    float3 *means = new float3[num_points];
    float3 *scales = new float3[num_points];
    float4 *quats = new float4[num_points];
    float3 *rgbs = new float3[num_points];
    float *opacities = new float[num_points];
    float viewmat [] = {
        1.f, 0.f, 0.f, 0.f,
        0.f, 1.f, 0.f, 0.f,
        0.f, 0.f, 1.f, 8.f,
        0.f, 0.f, 0.f, 1.f
    };

    // silly initialization of gaussians
    for (int i = 0; i < num_points; ++i) {
        float v = (float) i - (float) num_points * 0.5f;
        means[i] = {v * 0.2f, v * 0.2f, (float) i};
        scales[i] = {1.f, 2.f, 3.f};
        quats[i] = {1.f, 0.f, 0.f, 0.f};  // w x y z convention
        rgbs[i] = {1.f, 1.f, 1.f};
        opacities[i] = 0.8f;
    }

    float3 *scales_d, *means_d, *rgbs_d;
    float4 *quats_d;
    float *viewmat_d, *opacities_d;

    hipMalloc((void**) &scales_d, num_points * sizeof(float3));
    hipMalloc((void**) &means_d, num_points * sizeof(float3));
    hipMalloc((void**) &quats_d, num_points * sizeof(float4));
    hipMalloc((void**) &rgbs_d, num_points * sizeof(float3));
    hipMalloc((void**) &opacities_d, num_points * sizeof(float));
    hipMalloc((void**) &viewmat_d, num_view * sizeof(float));

    hipMemcpy(scales_d, scales, num_points * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(means_d, means, num_points * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(rgbs_d, rgbs, num_points * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(opacities_d, opacities, num_points * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(quats_d, quats, num_points * sizeof(float4), hipMemcpyHostToDevice);
    hipMemcpy(viewmat_d, viewmat, num_view * sizeof(float), hipMemcpyHostToDevice);

    // allocate memory for outputs
    float *covs3d = new float[num_cov3d];
    float2 *xy = new float2[num_points];
    float *z = new float[num_points];
    int *radii = new int[num_points];
    float3 *conics = new float3[num_points];
    uint32_t *num_tiles_hit = new uint32_t[num_points];

    float *covs3d_d, *z_d;
    float2 *xy_d;
    float3 *conics_d;
    int *radii_d;
    uint32_t *num_tiles_hit_d;
    hipMalloc((void**)&covs3d_d, num_cov3d * sizeof(float));
    hipMalloc((void**)&xy_d, num_points * sizeof(float2));
    hipMalloc((void**)&z_d, num_points * sizeof(float));
    hipMalloc((void**)&radii_d, num_points * sizeof(int));
    hipMalloc((void**)&conics_d, num_points * sizeof(float3));
    hipMalloc((void**)&num_tiles_hit_d, num_points * sizeof(uint32_t));

    project_gaussians_forward_impl(
        num_points,
        means_d,
        scales_d,
        1.f,
        quats_d,
        viewmat_d,
        viewmat_d,
        focal,
        focal,
        W,
        H,
        tile_bounds,
        covs3d_d,
        xy_d,
        z_d,
        radii_d,
        conics_d,
        num_tiles_hit_d
    );
    hipMemcpy(covs3d, covs3d_d, num_cov3d * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(xy, xy_d, num_points * sizeof(float2), hipMemcpyDeviceToHost);
    hipMemcpy(z, z_d, num_points * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(radii, radii_d, num_points * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(num_tiles_hit, num_tiles_hit_d, num_points * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // for (int i = 0; i < num_points; ++i) {
    //     printf("covs3d %d ", i);
    //     for (int j=0; j < 6; ++j) {
    //         printf("%.2f,", covs3d[6*i+j]);
    //     }
    //     printf("\n");
    // }

    uint32_t num_intersects;
    uint32_t *cum_tiles_hit = new uint32_t[num_points];
    uint32_t *cum_tiles_hit_d;
    hipMalloc((void**) &cum_tiles_hit_d, num_points * sizeof(uint32_t));
    compute_cumulative_intersects(num_points, num_tiles_hit_d, num_intersects, cum_tiles_hit_d);
    printf("num_intersects %d\n", num_intersects);
    hipMemcpy(cum_tiles_hit, cum_tiles_hit_d, num_points * sizeof(uint32_t), hipMemcpyDeviceToHost);
    for (int i = 0; i < num_points; ++i) {
        printf("cum_tiles_hit %d, %d\n", i, cum_tiles_hit[i]);
    }

    uint32_t *gaussian_ids_sorted_d;  // sorted by tile and depth
    uint32_t *gaussian_ids_sorted = new uint32_t[num_intersects];

    int num_tiles = tile_bounds.x * tile_bounds.y;
    uint2 *tile_bins_d;  // start and end indices for each tile
    uint2 *tile_bins = new uint2[num_tiles];
    hipMalloc((void**) &gaussian_ids_sorted_d, num_intersects * sizeof(uint32_t));
    hipMalloc((void**) &tile_bins_d, num_tiles * sizeof(uint2));

    bin_and_sort_gaussians(
        num_points,
        num_intersects,
        xy_d,
        z_d,
        radii_d,
        cum_tiles_hit_d,
        tile_bounds,
        gaussian_ids_sorted_d,
        tile_bins_d
    );
    hipMemcpy(gaussian_ids_sorted, gaussian_ids_sorted_d, num_intersects * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(tile_bins, tile_bins_d, num_tiles * sizeof(uint2), hipMemcpyDeviceToHost);
    for (int i = 0; i < num_intersects; ++i) {
        printf("gaussian_id %d %d\n", i, gaussian_ids_sorted[i]);
    }
    for (int i = 0; i < num_tiles; ++i) {
        printf("tile_bins %d %d %d\n", i, tile_bins[i].x, tile_bins[i].y);
    }

    float3 *out_img = new float3[W * H];
    float3 *out_img_d;
    hipMalloc((void**)&out_img_d, W * H * sizeof(float3));

    rasterize_forward_impl(
        tile_bounds,
        block,
        img_size,
        gaussian_ids_sorted_d,
        tile_bins_d,
        xy_d,
        conics_d,
        rgbs_d,
        opacities_d,
        out_img_d
    );
    hipMemcpy(out_img, out_img_d, W * H * sizeof(float3), hipMemcpyDeviceToHost);
    TGAImage image(W, H, TGAImage::RGB);
    int idx;
    float3 c;
    TGAColor col;
    for (int y = 0; y < H; ++y) {
        for (int x = 0; x < W; ++x) {
            idx = y * W + x;
            c = out_img[idx];
            col[0] = (uint8_t) (255.f * c.x);
            col[1] = (uint8_t) (255.f * c.y);
            col[2] = (uint8_t) (255.f * c.z);
            col[3] = 255;
            image.set(x, y, col);
        }
    }
    image.write_tga_file("output.tga");

    printf("freeing memory...\n");

    hipFree(scales_d);
    hipFree(quats_d);
    hipFree(rgbs_d);
    hipFree(opacities_d);
    hipFree(covs3d_d);
    hipFree(viewmat_d);
    hipFree(xy_d);
    hipFree(z_d);
    hipFree(radii_d);
    hipFree(num_tiles_hit_d);
    hipFree(cum_tiles_hit_d);
    hipFree(tile_bins_d);
    hipFree(gaussian_ids_sorted_d);
    hipFree(conics_d);
    hipFree(out_img_d);

    delete[] scales;
    delete[] means;
    delete[] rgbs;
    delete[] opacities;
    delete[] quats;
    delete[] covs3d;
    delete[] xy;
    delete[] z;
    delete[] radii;
    delete[] num_tiles_hit;
    delete[] cum_tiles_hit;
    delete[] tile_bins;
    delete[] gaussian_ids_sorted;
    delete[] out_img;
    return 0;
}
