#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <math.h>

#include "forward.cuh"


int main() {
    int num_points = 24;
    const float fov_x = M_PI / 2.f;
    const int W = 128;
    const int H = 128;
    const float focal = 0.5 * (float) W / tan(0.5 * fov_x);
    const dim3 tile_bounds = {(W + BLOCK_X - 1) / BLOCK_X, (H + BLOCK_Y - 1) / BLOCK_Y, 1};

    int num_mean = num_points * 3;
    int num_scale = num_points * 3;
    int num_quat = num_points * 4;
    int num_cov3d = num_points * 6;
    int num_xy = num_points * 2;
    int num_z = num_points;
    int num_radii = num_points;
    int num_view = 16;

    float *means = new float[num_mean];
    float *scales = new float[num_scale];
    float *quats = new float[num_quat];
    float *covs3d = new float[num_cov3d];
    float *xy = new float[num_xy];
    float *z = new float[num_z];
    int *radii = new int[num_radii];
    uint32_t *num_tiles_hit = new uint32_t[num_radii];
    float viewmat [] = {
        1.f, 0.f, 0.f, 0.f,
        0.f, 1.f, 0.f, 0.f,
        0.f, 0.f, 1.f, 8.f,
        0.f, 0.f, 0.f, 1.f
    };

    for (int i = 0; i < num_points; ++i) {
        float x [] = {(float) i * 0.1f, (float) i * 0.1f, (float) i};
        float s [] = {(float) i * 0.5f, (float) i * 0.5f, (float) i * 0.5f};
        float q [] = {1.f, 0.f, 0.f, 0.f};
        std::memcpy(&means[3 * i], &x, sizeof(float) * 3);
        std::memcpy(&scales[3 * i], &s, sizeof(float) * 3);
        std::memcpy(&quats[4 * i], &q, sizeof(float) * 4);
        // printf("scales %d, %.2f, %.2f, %.2f\n", i, scales[3*i], scales[3*i+1], scales[3*i+2]);
        // printf("quats %d, %.2f, %.2f, %.2f, %.2f\n", i, quats[4*i], quats[4*i+1], quats[4*i+2], quats[4*i+3]);
    }

    float *scales_d, *means_d, *quats_d, *viewmat_d;

    hipMalloc((void**) &scales_d, num_scale * sizeof(float));
    hipMalloc((void**) &means_d, num_mean * sizeof(float));
    hipMalloc((void**) &quats_d, num_quat * sizeof(float));
    hipMalloc((void**) &viewmat_d, num_view * sizeof(float));

    hipMemcpy(scales_d, scales, num_scale * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(means_d, means, num_mean * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(quats_d, quats, num_quat * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(viewmat_d, viewmat, num_view * sizeof(float), hipMemcpyHostToDevice);

    // allocate memory for outputs
    float *covs3d_d, *xy_d, *z_d;
    int *radii_d;
    uint32_t *num_tiles_hit_d;
    hipMalloc((void**)&covs3d_d, num_cov3d * sizeof(float));
    hipMalloc((void**)&xy_d, num_xy * sizeof(float));
    hipMalloc((void**)&z_d, num_z * sizeof(float));
    hipMalloc((void**)&radii_d, num_radii * sizeof(int));
    hipMalloc((void**)&num_tiles_hit_d, num_radii * sizeof(uint32_t));

    project_gaussians_forward_impl(
        num_points,
        means_d,
        scales_d,
        1.f,
        quats_d,
        viewmat_d,
        viewmat_d,
        focal,
        focal,
        W,
        H,
        tile_bounds,
        covs3d_d,
        xy_d,
        z_d,
        radii_d,
        num_tiles_hit_d
    );
    hipMemcpy(covs3d, covs3d_d, num_cov3d * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(xy, xy_d, num_xy * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(z, z_d, num_z * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(radii, radii_d, num_radii * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(num_tiles_hit, num_tiles_hit_d, num_radii * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // for (int i = 0; i < num_points; ++i) {
    //     printf("covs3d %d ", i);
    //     for (int j=0; j < 6; ++j) {
    //         printf("%.2f,", covs3d[6*i+j]);
    //     }
    //     printf("\n");
    //
    //     printf("xy %d ", i);
    //     for (int j=0; j < 3; ++j) {
    //         printf("%.2f,", xy[3*i+j]);
    //     }
    //     printf("\n");
    // }

    uint32_t num_intersects;
    uint32_t *cum_tiles_hit = new uint32_t[num_points];
    uint32_t *cum_tiles_hit_d;
    hipMalloc((void**) &cum_tiles_hit_d, num_points * sizeof(uint32_t));
    compute_cumulative_intersects(num_points, num_tiles_hit_d, num_intersects, cum_tiles_hit_d);
    printf("num_intersects %d\n", num_intersects);
    hipMemcpy(cum_tiles_hit, cum_tiles_hit_d, num_points * sizeof(uint32_t), hipMemcpyDeviceToHost);
    for (int i = 0; i < num_points; ++i) {
        printf("cum_tiles_hit %d, %d\n", i, cum_tiles_hit[i]);
    }

    uint32_t *gaussian_ids_sorted_d;  // sorted by tile and depth
    uint32_t *gaussian_ids_sorted = new uint32_t[num_intersects];

    int num_tiles = tile_bounds.x * tile_bounds.y;
    uint2 *tile_bins_d;  // start and end indices for each tile
    uint2 *tile_bins = new uint2[num_tiles];
    hipMalloc((void**) &gaussian_ids_sorted_d, num_intersects * sizeof(uint32_t));
    hipMalloc((void**) &tile_bins_d, num_tiles * sizeof(uint2));

    bin_and_sort_gaussians(
        num_points,
        num_intersects,
        xy_d, 
        z_d,
        radii_d,
        cum_tiles_hit_d,
        tile_bounds,
        gaussian_ids_sorted_d,
        tile_bins_d
    );
    hipMemcpy(gaussian_ids_sorted, gaussian_ids_sorted_d, num_intersects * sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(tile_bins, tile_bins_d, num_tiles * sizeof(uint2), hipMemcpyDeviceToHost);
    for (int i = 0; i < num_intersects; ++i) {
        printf("gaussian_id %d %d\n", i, gaussian_ids_sorted[i]);
    }
    for (int i = 0; i < num_tiles; ++i) {
        printf("tile_bins %d %d %d\n", i, tile_bins[i].x, tile_bins[i].y);
    }

    printf("freeing memory...\n");

    hipFree(scales_d);
    hipFree(quats_d);
    hipFree(covs3d_d);
    hipFree(viewmat_d);
    hipFree(xy_d);
    hipFree(z_d);
    hipFree(radii_d);
    hipFree(num_tiles_hit_d);
    hipFree(cum_tiles_hit_d);
    hipFree(tile_bins_d);
    hipFree(gaussian_ids_sorted_d);

    delete[] scales;
    delete[] means;
    delete[] quats;
    delete[] covs3d;
    delete[] xy;
    delete[] z;
    delete[] radii;
    delete[] num_tiles_hit;
    delete[] cum_tiles_hit;
    delete[] tile_bins;
    delete[] gaussian_ids_sorted;
    return 0;
}
