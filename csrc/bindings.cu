#include "hip/hip_runtime.h"
#include <math.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <tuple>
#include <hip/hip_runtime_api.h>
#include "helpers.cuh"
#include "bindings.h"
#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>

namespace cg = cooperative_groups;


template <typename scalar_t>
__global__ void compute_cov2d_bounds_kernel_forward(
    const int num_pts,
    const scalar_t * __restrict__ A,
    scalar_t * __restrict__ conics,
    scalar_t * __restrict__ radii
){   
    unsigned row = cg::this_grid().thread_rank(); // same as threadIdx.x + blockIdx.x * blockDim.x;
    if (row>=num_pts){return;}
    int index = row * 3;

    float3 conic;
    float radius;
    float3 cov2d{(float)A[index], (float)A[index+1], (float)A[index+2]};
    compute_cov2d_bounds(cov2d,conic,radius);
    
    conics[index] = conic.x;
    conics[index+1] = conic.y;
    conics[index+2] = conic.z;
    radii[row]=radius;
}


std::tuple<
    torch::Tensor, // output conics
    torch::Tensor // ouptut radii
    >
compute_cov2d_bounds_cu_forward(
    int num_pts,
    torch::Tensor A
){

    torch::Tensor conics = torch::zeros({num_pts, A.size(1)}, A.options().dtype(torch::kFloat32));
    torch::Tensor radii = torch::zeros({num_pts, 1}, A.options().dtype(torch::kFloat32));

    const int threads = 256;
    int blocks = (num_pts + threads - 1) / threads;
    // instantiate kernel
    AT_DISPATCH_FLOATING_TYPES(A.type(), "compute_cov2d_bounds_cu_forward", 
    ([&] {
        compute_cov2d_bounds_kernel_forward<scalar_t><<<blocks, threads>>>(
            num_pts,
            A.contiguous().data_ptr<scalar_t>(),
            conics.contiguous().data_ptr<scalar_t>(),
            radii.contiguous().data_ptr<scalar_t>()
        );
    })
    );
    return std::make_tuple(conics, radii);
}

std::tuple<
    torch::Tensor, // output conics
    torch::Tensor // output radii
    >
compute_cov2d_bounds_forward(
    const int num_pts,
    torch::Tensor A)
{
    CHECK_INPUT(A);
    return compute_cov2d_bounds_cu_forward(num_pts, A);
}