#include "hip/hip_runtime.h"
#include "bindings.h"
#include "forward.cuh"
#include "helpers.cuh"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <iostream>
#include <math.h>
#include <torch/extension.h>
#include <tuple>

namespace cg = cooperative_groups;

template <typename scalar_t>
__global__ void compute_cov2d_bounds_forward_kernel(
    const int num_pts,
    const scalar_t *__restrict__ A,
    scalar_t *__restrict__ conics,
    scalar_t *__restrict__ radii
) {
    unsigned row = cg::this_grid().thread_rank(
    ); // same as threadIdx.x + blockIdx.x * blockDim.x;
    if (row >= num_pts) {
        return;
    }
    int index = row * 3;

    float3 conic;
    float radius;
    float3 cov2d{(float)A[index], (float)A[index + 1], (float)A[index + 2]};
    compute_cov2d_bounds(cov2d, conic, radius);

    conics[index] = conic.x;
    conics[index + 1] = conic.y;
    conics[index + 2] = conic.z;
    radii[row] = radius;
}

std::
    tuple<
        torch::Tensor, // output conics
        torch::Tensor  // ouptut radii
        >
    compute_cov2d_bounds_forward_tensor(const int num_pts, torch::Tensor A) {
    CHECK_INPUT(A);

    torch::Tensor conics =
        torch::zeros({num_pts, A.size(1)}, A.options().dtype(torch::kFloat32));
    torch::Tensor radii =
        torch::zeros({num_pts, 1}, A.options().dtype(torch::kFloat32));

    int blocks = (num_pts + N_THREADS - 1) / N_THREADS;
    // instantiate kernel
    AT_DISPATCH_FLOATING_TYPES(
        A.type(), "compute_cov2d_bounds_cu_forward", ([&] {
            compute_cov2d_bounds_forward_kernel<scalar_t>
                <<<blocks, N_THREADS>>>(
                    num_pts,
                    A.contiguous().data_ptr<scalar_t>(),
                    conics.contiguous().data_ptr<scalar_t>(),
                    radii.contiguous().data_ptr<scalar_t>()
                );
        })
    );
    return std::make_tuple(conics, radii);
}
