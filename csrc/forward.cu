#include "hip/hip_runtime.h"
#include "forward.h"
#include <iostream>
#include <glm/glm.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;


// host function to launch the projection in parallel on device
void project_gaussians_forward_impl(
    const int num_points,
    const float *means3d,
    const float *scales,
    const float glob_scale,
    const float *rots_quat,
    const float *viewmat,
    float *covs3d,
    float *covs2d
) {
    int num_threads = 16;
    project_gaussians_forward_kernel
    <<< (num_points + num_threads - 1) / num_threads, num_threads >>> (
        num_points,
        means3d,
        scales,
        glob_scale,
        rots_quat,
        viewmat,
        covs3d,
        covs2d
    );
}


// kernel function for projecting each gaussian on device
// each thread processes one gaussian
__global__ void project_gaussians_forward_kernel(
    const int num_points,
    const float *means3d,
    const float *scales,
    const float glob_scale,
    const float *rots_quat,
    const float *viewmat,
    float *covs3d,
    float *covs2d
) {
    unsigned idx = cg::this_grid().thread_rank();  // idx of thread within grid
    if (idx >= num_points) {
        return;
    }
    float3 p_world = {means3d[3*idx], means3d[3*idx+1], means3d[3*idx+2]};
    // printf("hello %d", idx);
    // printf("%.2f %.2f %.2f\n", scales[0], scales[1], scales[2]);
    float3 scale = {scales[3*idx], scales[3*idx+1], scales[3*idx+2]};
    float4 quat = {rots_quat[4*idx+1], rots_quat[4*idx +2], rots_quat[4*idx + 3], rots_quat[4*idx]};
    // printf("0 scale %.2f %.2f %.2f\n", scale.x, scale.y, scale.z);
    // printf("0 quat %.2f %.2f %.2f %.2f\n", quat.w, quat.x, quat.y, quat.z);
    compute_cov3d(scale, glob_scale, quat, &(covs3d[6 * idx]));
    project_cov3d_ewa(p_world, &(covs3d[6*idx]), viewmat, 1.f, 1.f, &(covs2d[3*idx]));
}


// host function to launch parallel rendering of sorted gaussians on device
void render_forward_impl(
) {
}


// kernel function for rendering each gaussian on device
__global__ void render_forward_kernel(
) {
}


// device helper to approximate projected 2d cov from 3d mean and cov
__device__ void project_cov3d_ewa(
    const float3 &mean3d,
    const float *cov3d,
    const float *viewmat,
    const float fx,
    const float fy,
    // const float tan_fovx,
    // const float tan_fovy,
    float *cov2d
) {
    // we expect row major matrices as input,
    // glm uses column major
    glm::mat4 viewmat_glm = glm::transpose(glm::make_mat4(viewmat));
    glm::vec4 t = viewmat_glm * glm::vec4(mean3d.x, mean3d.y, mean3d.z, 1.f);
    // printf("viewmat_glm %.2f %.2f %.2f %.2f\n", viewmat_glm[0][0], viewmat_glm[1][1], viewmat_glm[2][2], viewmat_glm[3][3]);
    // printf("t %.2f %.2f %.2f %.2f\n", t[0], t[1], t[2], t[3]);
    // printf("t %.2f %.2f %.2f %.2f\n", t.w, t.x, t.y, t.z);

    // column major
    glm::mat3 J = glm::transpose(glm::mat3(
        fx / t.z, 0.f, -fx * t.x / (t.z * t.z),
        0.f, fy / t.z, -fy * t.y / (t.z * t.z),
        0.f, 0.f, 0.f
    ));
    glm::mat3 W = glm::mat3(
        glm::vec3(viewmat_glm[0]), glm::vec3(viewmat_glm[1]), glm::vec3(viewmat_glm[2])
    );

    glm::mat3 T = J * W;

    glm::mat3 V = glm::mat3(
        cov3d[0], cov3d[1], cov3d[2],
        cov3d[1], cov3d[3], cov3d[4],
        cov3d[2], cov3d[4], cov3d[5]
    );

    // printf("J %.2f %.2f %.2f\n", J[0][0], J[1][1], J[2][2]);
    // printf("W %.2f %.2f %.2f\n", W[0][0], W[1][1], W[2][2]);
    // printf("V %.2f %.2f %.2f\n", V[0][0], V[1][1], V[2][2]);

    // we only care about the top 2x2 submatrix
    glm::mat3 cov = T * V * glm::transpose(T);
    // add a little blur along axes and save upper triangular elements
    cov2d[0] = float(cov[0][0]) + 0.1f;
    cov2d[1] = float(cov[0][1]);
    cov2d[2] = float(cov[1][1]) + 0.1f;
}


// device helper to get 3D covariance from scale and quat parameters
__device__ void compute_cov3d(
    const float3 scale, const float glob_scale, const float4 quat, float *cov3d
) {
    // printf("scale %.2f %.2f %.2f\n", scale.x, scale.y, scale.z);
    // printf("quat %.2f %.2f %.2f %.2f\n", quat.w, quat.x, quat.y, quat.z);
    // quat to rotation matrix
    float s = rsqrtf(
        quat.w * quat.w + quat.x * quat.x + quat.y * quat.y + quat.z * quat.z
    );
    float w = quat.w * s;
    float x = quat.x * s;
    float y = quat.y * s;
    float z = quat.z * s;

    // glm matrices are column-major
    glm::mat3 R = glm::mat3(
        1.f - 2.f * (y * y + z * z), 2.f * (x * y + w * z), 2.f * (x * z - w * y),
        2.f * (x * y - w * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z + w * x),
        2.f * (x * z + w * y), 2.f * (y * z - w * x), 1.f - 2.f * (x * x + x * x)
    );
    // printf("R %.2f %.2f %.2f\n", R[0][0], R[1][1], R[2][2]);

    glm::mat3 S = glm::mat3(1.f);
    S[0][0] = glob_scale * scale.x;
    S[1][1] = glob_scale * scale.y;
    S[2][2] = glob_scale * scale.z;
    // printf("S %.2f %.2f %.2f\n", S[0][0], S[1][1], S[2][2]);

    glm::mat3 M = R * S;
    glm::mat tmp = M * glm::transpose(M);
    // printf("tmp %.2f %.2f %.2f\n", tmp[0][0], tmp[1][1], tmp[2][2]);

    // save upper right because symmetric
    cov3d[0] = tmp[0][0];
    cov3d[1] = tmp[0][1];
    cov3d[2] = tmp[0][2];
    cov3d[3] = tmp[1][1];
    cov3d[4] = tmp[1][2];
    cov3d[5] = tmp[2][2];
}
